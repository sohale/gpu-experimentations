#include "hip/hip_runtime.h"
#include "profilers.h"

#include "runtime_profiling_reporter.h"

#include <cstdlib>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

__global__ void matrixMultiplyNaive(float *W, float *X, float *Y, int n, int m);

#define M 25600 // Number of input vectors in a batch
// #define N 1024 // Matrix dimension (adjust as needed)

// stores
InMemoryStructuredReporter reporter;

// measures time
Profiler profiler;

void executeTrial(float *d_W, float *d_X, float *d_Y, float *h_W, float *h_X,
                  float *h_Y, int N, int Nrep, int t) {
  auto s = profiler.start();

  // Copy data from host to device
  hipMemcpy(d_W, h_W, N * N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_X, h_X, N * M * sizeof(float), hipMemcpyHostToDevice);

  // Define grid and block sizes
  dim3 blockSize(16, 16);
  dim3 gridSize((M + blockSize.x - 1) / blockSize.x,
                (N + blockSize.y - 1) / blockSize.y);

  for (int r = 0; r < Nrep; ++r) {
    // Launch naive matrix multiplication kernel
    matrixMultiplyNaive<<<gridSize, blockSize>>>(d_W, d_X, d_Y, N, M);
  }

  // Copy result back to host
  hipMemcpy(h_Y, d_Y, N * M * sizeof(float), hipMemcpyDeviceToHost);

  double elapsed = s.stop();

  // reporter.report_measurement(N, Nrep, t, elapsed.count() );
  reporter.report_measurement(
      InMemoryStructuredReporter::ProfilingEntry{N, M, Nrep, t, elapsed});
}

void runExperiment(int N, int Nrep, int Ntrials) {
  // Allocate memory on host
  float *h_W = (float *)malloc(N * N * sizeof(float));
  float *h_X = (float *)malloc(N * M * sizeof(float));
  float *h_Y = (float *)malloc(N * M * sizeof(float));

  // Initialize W and X with random values
  for (int i = 0; i < N * N; ++i)
    h_W[i] = static_cast<float>(rand()) / RAND_MAX;
  for (int i = 0; i < N * M; ++i)
    h_X[i] = static_cast<float>(rand()) / RAND_MAX;

  // Allocate memory on device
  float *d_W, *d_X, *d_Y;
  hipMalloc((void **)&d_W, N * N * sizeof(float));
  hipMalloc((void **)&d_X, N * M * sizeof(float));
  hipMalloc((void **)&d_Y, N * M * sizeof(float));

  for (int t = 0; t < Ntrials; ++t) {
    executeTrial(d_W, d_X, d_Y, h_W, h_X, h_Y, N, Nrep, t);
  }

  // Cleanup
  hipFree(d_W);
  hipFree(d_X);
  hipFree(d_Y);
  free(h_W);
  free(h_X);
  free(h_Y);
}

void runProfiling(std::vector<int> N_k, int Nrep, int Ntrials) {
  for (int N : N_k) {
    runExperiment(N, Nrep, Ntrials);
  }
}

int main() {
  std::vector<int> N_k = {256, 512, 1024, 2048}; // Example sequence of N values
  int Nrep = 10;   // Number of kernel executions per measurement (once data is transferred into (device) GPU global memory)
  // this is separated, to separate the time taken to transfer data between CPU memory & GPU global memory,
  // with the time taken to execute the kernel.
  int Ntrials = 5; // Number of repeated measurements per N (includes transfer time)
  // M: batch size (one trnsaciton, whole calculation.).
  // Matrix A is of size N x N,
  // matrix B is of size N x M.

  std::cout << "This may take a while, please wait..." << std::endl;
  runProfiling(N_k, Nrep, Ntrials);

  std::cout << "Profiling completed." << std::endl;
  return 0;
}
