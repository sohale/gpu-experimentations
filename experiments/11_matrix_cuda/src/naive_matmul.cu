// #include <cstdlib>
#include <hip/hip_runtime.h>

/*
Notes:
    batch (ML lingo) = block of stream (CE lingo)
*/

__global__ void matrixMultiplyNaive(float *W, float *X, float *Y, int n,
                                    int m) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < n && col < m) {
    float sum = 0.0f;
    for (int k = 0; k < n; ++k) {
      sum += W[row * n + k] * X[k * m + col];
    }
    Y[row * m + col] = sum;
  }
}
